#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

__host__
void fill_matrix(float* M , int row , int col){
	for (int i = 0; i < row; ++i){
		for (int j = 0; j < col; ++j)
		{
			M[i*col + j] = 2.0;
		}
	}
}

__host__
void read(float *M, FILE *source, int rows, int cols){
	for (int i = 0; i < rows; ++i){
		for (int j = 0; j < cols; ++j){
			fscanf(source, "%f,", &M[i * cols + j]);
		}
	}
	fclose(source);
	return;
}

__host__
void print(float *M, int rows, int cols){
  printf("\n");
  printf("----------------------------------------\n");
  for(int i = 0; i < rows; i++) {
  		for(int j = 0; j < cols; j++) {
     		printf("%.2f ", M[i * cols + j]);
    	}
		printf("\n");
  }
  printf("----------------------------------------\n");
  printf("\n");
  return;
}

__global__
void MatrixMultiplySMKernel(float *d_A, float *d_B, float *d_R, int colsA, int rowsA, int colsB, int rowsB){

	__shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;


	int col = bx * TILE_WIDTH + tx;
	int row = by * TILE_WIDTH + ty;

	float Pvalue = 0;
	for (int m = 0; m < (TILE_WIDTH + colsA - 1)/TILE_WIDTH; ++m){

		if(m * TILE_WIDTH + tx < colsA && row < rowsA){
			Ads[ty][tx] = d_A[row * colsA + m * TILE_WIDTH + tx];
		}
		else{
			Ads[ty][tx] = 0.0;
		}

		if(m * TILE_WIDTH + ty < rowsB && col < colsB){
			Bds[ty][tx] = d_B[(m * TILE_WIDTH + ty) * colsB + col];
		}
		else{
			Bds[ty][tx] = 0.0;
		}

		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k){
			Pvalue += Ads[ty][k] * Bds[k][tx];
		}

		__syncthreads();
	}

	if(row < rowsA && col < colsB){
		d_R[((by * blockDim.y + ty) * colsB) + (bx * blockDim.x) + tx] = Pvalue;
	}
	return;
}


int main(int argc, char** argv)
{

	/*if (argc != 3){
		printf("Debe añadir los nombres de los archivos\n");
		return 1;
	}*/

	float *h_A, *h_B, *h_R;
	int rowsA, rowsB, colsA, colsB = 100;



	hipError_t error = hipSuccess;

	//FILE *file_1, *file_2;
	//file_1 = fopen(argv[1], "r");
	//file_2 = fopen(argv[2], "r");

	//fscanf(file_1, "%d", &rowsA);
	//fscanf(file_1, "%d", &colsA);
	//fscanf(file_2, "%d", &rowsB);
	//fscanf(file_2, "%d", &colsB);

	if (colsA != rowsB){
		printf("Es imposible multiplicar las matrices\n");
		return 1;
	}

	float sizeA = rowsA * colsA * sizeof(float);
	float sizeB = rowsB * colsB * sizeof(float);
	float sizeR = rowsA * colsB * sizeof(float);


	h_A = (float*)malloc(sizeA);
	h_B = (float*)malloc(sizeB);
	h_R = (float*)malloc(sizeR);

	//read(h_A, file_1, rowsA, colsA);
	//read(h_B, file_2, rowsB, colsB);

	fill_matrix(*h_A, rowsA, colsA);
	fill_matrix(*h_B, rowsB, colsB);

	float *d_A, *d_B, *d_R;

	error = hipMalloc((void**)&d_A, sizeA);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_A \n");
		return 1;
	}

	error = hipMalloc((void**)&d_B, sizeB);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_B \n");
		return 1;
	}

	error = hipMalloc((void**)&d_R, sizeR);
	if (error != hipSuccess){
		printf("Error solicitando memoria para d_R \n");
		return 1;
	}

	hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

	int blockSize = 32;
	dim3 dimGrid(32, 32, 1);
	dim3 dimBlock(blockSize, blockSize, 1);

	MatrixMultiplySMKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_R, colsA, rowsA, colsB, rowsB);
	hipMemcpy(h_R, d_R, sizeR, hipMemcpyDeviceToHost);

	print(h_A, rowsA, colsA);
	print(h_B, rowsB, colsB);
	print(h_R, rowsA, colsB);


	free(h_A);
	free(h_B);
	free(h_R);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_R);


	/* code */
	return 0;
}
