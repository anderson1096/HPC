#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



__host__
void fill_vector(float *V, int len){
  float aux = 5.0;
  for (int i = 0; i < len; i++) {
    V[i] = ((float)rand() / (float)(RAND_MAX)) * aux ;
  }
}

__host__
void print(float *V, int len){
  for (int i = 0; i < len; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__
void MatrixKernel(float* d_M, float* d_R, int n){
  //calculate row index of element
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < n) d_R[i] = 2 * d_M[i];
  return;
}


int main(){
  int n = 100;
  int size = n * sizeof(float);

  //Manejo de errores en cuda
  hipError_t error = hipSuccess;

  //CPU
  float *h_M, *h_R;
  h_M = (float*)malloc(size);
  h_R = (float*)malloc(size);
  

  //GPU
  float *d_M, *d_R;
  
  error = hipMalloc((void**)&d_M, size);
  if (error != hipSuccess){
    printf("Error solicitando memoria en la GPU para d_M\n");
    exit(-1);
  }

  error = hipMalloc((void**)&d_R, size);
  if (error != hipSuccess){
    printf("Error solicitando memoria en la GPU para d_R\n");
    exit(-1);
  }

  //Fill Matrix
  fill_vector(h_M, size);

  //Copy from CPU to GPU
  hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

  //Dimension kernel
  dim3 dimGrid(ceil(n/10.0), 1, 1);
  dim3 dimBlock(10,1,1);
  MatrixKernel<<<dimGrid, dimBlock>>>(d_M, d_R, n);
  hipDeviceSynchronize();

  
  hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
  print(h_R, n);


  hipFree(d_M);
  hipFree(d_R);
  free(h_M);
  free(h_R);
  
  return 0;
}
