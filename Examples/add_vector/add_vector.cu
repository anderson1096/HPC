#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>



__host__
void fill_vector(float *V, int len){
  float aux = 5.0;
  for (int i = 0; i < len; i++) {
    V[i] = ((float)rand() / (float)(RAND_MAX)) * aux ;
  }
}

__host__
void print(float *V, int len){
  for (int i = 0; i < len; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__
void AddVector(float* d_A, float* d_B, float* d_R, int n){
  //calculate row index of element
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < n) d_R[i] = d_A[i] + d_B[i];
  return;
}


int main(){
  int n = 100;
  float size = n * sizeof(float);

  //Manejo de errores en cuda
  hipError_t error = hipSuccess;

  //CPU
  float *h_A, *h_B, *h_R;
  h_A = (float*)malloc(size);
  h_B = (float*)malloc(size);
  h_R = (float*)malloc(size);


  //GPU
  float *d_A, *d_B, *d_R;

  error = hipMalloc((void**)&d_A, size);
  if (error != hipSuccess){
    printf("Error solicitando memoria en la GPU para d_A\n");
    exit(-1);
  }

  error = hipMalloc((void**)&d_B, size);
  if (error != hipSuccess){
    printf("Error solicitando memoria en la GPU para d_B\n");
    exit(-1);
  }

  error = hipMalloc((void**)&d_R, size);
  if (error != hipSuccess){
    printf("Error solicitando memoria en la GPU para d_R\n");
    exit(-1);
  }

  //Fill Matrix
  fill_vector(h_A, n);
  fill_vector(h_B, n);
  print(h_A, n);
  printf("---------------------------------\n");
  print(h_B, n);
  printf("---------------------------------\n");
  //Copy from CPU to GPU
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  //Dimension kernel
  dim3 dimGrid(ceil(n/10.0), 1, 1);
  dim3 dimBlock(10,1,1);
  AddVector<<<dimGrid, dimBlock>>>(d_A, d_B, d_R, n);
  hipDeviceSynchronize();


  hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
  print(h_R, n);

  free(h_A); free(h_B); free(h_R);
  hipFree(d_A); hipFree(d_B); hipFree(d_R);

  return 0;
}
